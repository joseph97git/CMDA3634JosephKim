#include "hip/hip_runtime.h"
#include <std.io>
#include <stdlib.h>

#include "hip/hip_runtime.h"

// This is my DEVICE function 
// __global__ mean this function is visible to the host
__global__ void kernelHelloWorld() {
  
  print("Hello World!\n");

}

int main(int argc, char** argv) {

  int Nblocks = 10; //number of blocks
  int Nthreads = 3; //number of threads per block

  // run the function 'kernelHellowWorld' on the DEVICE
  kernelHelloWorld <<< Nblocks, Nthreads >>> ();


}
