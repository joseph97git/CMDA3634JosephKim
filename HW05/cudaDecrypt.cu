#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include "functions.c"


//device functions

//compute a*b mod p safely
__device__ unsigned int kernelModProd(unsigned int a, unsigned int b, unsigned int p) {
  unsigned int za = a;
  unsigned int ab = 0;

  while (b > 0) {
    if (b%2 == 1) ab = (ab +  za) % p;
    za = (2 * za) % p;
    b /= 2;
  }
  return ab;
}

//compute a^b mod p safely
__device__ unsigned int kernelModExp(unsigned int a, unsigned int b, unsigned int p) {
  unsigned int z = a;
  unsigned int aExpb = 1;

  while (b > 0) {
    if (b%2 == 1) aExpb = kernelModProd(aExpb, z, p);
    z = kernelModProd(z, z, p);
    b /= 2;
  }
  return aExpb;
}


__global__ void kernelFindKey(unsigned int n, unsigned int p,
							  unsigned int g, unsigned int h,
							  unsigned int* x) {
	
	int threadid = threadIdx.x; //thread number
	int blockid = blockIdx.x; //block number
	int Nblock = blockDim.x; //number of threads in a block
	
	int id = threadid + blockid*Nblock;
	
	
	// find the secret key 
      if (kernelModExp(g,id+1,p)==h) {
        printf("Secret key found! x = %u \n", id+1);
        *x=id+1;
      } 

 
}



int main (int argc, char **argv) {

  /* Part 2. Start this program by first copying the contents of the main function from 
     your completed decrypt.c main function. */
     
   //declare storage for an ElGamal cryptosytem
  unsigned int n, p, g, h, x;
  unsigned int Nints;

  //get the secret key from the user
  printf("Enter the secret key (0 if unknown): "); fflush(stdout);
  char stat = scanf("%u",&x);

  printf("Reading file.\n");

  /* Q3 Complete this function. Read in the public key data from public_key.txt
    and the cyphertexts from messages.txt. */
    
  FILE * f;
  f = fopen("public_key.txt","r");
  
  // read in n,p,g,h
  fscanf(f,"%d",&n);
  fscanf(f,"%d",&p);
  fscanf(f,"%d",&g);
  fscanf(f,"%d",&h);
  fclose(f);
  
  FILE * of;
  of = fopen("message.txt","r");
  
  // read in Nints;
  fscanf(of,"%d",&Nints);

  
  /* Q4 Make the search for the secret key parallel on the GPU using CUDA. */

  //allocate storage on host
  unsigned int *h_result = (unsigned int *) malloc(sizeof(unsigned int));
 
  h_result[0] = x; //define h_result as the 
  
  //allocate storage on device
  unsigned int *d_result;
  hipMalloc(&d_result, sizeof(unsigned int));
  
  //define thread and block size
  int Nthreads = 64;
  int Nblocks = (p+Nthreads-1)/Nthreads;
  
   if (x==0 || modExp(g,x,p)!=h) {
	   
	
    double startTime = clock();  
    printf("Finding the secret key...\n");
    kernelFindKey <<<Nblocks,Nthreads>>>(n,p,g,h,d_result);
  
    hipDeviceSynchronize(); 
    double endTime = clock();
    
    double totalTime = (endTime-startTime)/CLOCKS_PER_SEC;
    double work = (double) p;
    double throughput = work/totalTime;
    
    printf("Searching all keys took %g seconds, throughput was %g values tested per second.\n", totalTime, throughput);
  }  
  //copy answer from device back to the host
  hipMemcpy(h_result,d_result,sizeof(double),hipMemcpyDeviceToHost);
  x = h_result[0]; //redefine x 
  
  
  /* Q3 After finding the secret key, decrypt the message */
  
  //storage for message as elements of Z_p
  unsigned int *Zmessage = 
      (unsigned int *) malloc(Nints*sizeof(unsigned int)); 
  
  //storage for extra encryption coefficient 
  unsigned int *a = 
      (unsigned int *) malloc(Nints*sizeof(unsigned int)); 
      
  for (int i=0;i<Nints;i++) {
	  fscanf(of,"%d %d", &Zmessage[i], &a[i]);
  }
  fclose(of);
  
  ElGamalDecrypt(Zmessage,a,Nints,p,x);
  
  unsigned int charsPerInt = (n-1)/8;
  unsigned int Nchars = Nints * charsPerInt;
  
  int bufferSize = 1024;
  unsigned char *message = (unsigned char *) calloc(bufferSize,sizeof(unsigned char));
  
  convertZToString(Zmessage, Nints, message, Nchars);

  printf("Decrypted Message = \"%s\"\n", message);
  printf("\n");
  
  hipFree(d_result);
  free(h_result);
  
  return 0;
}
